#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_1D_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)


__global__ void build_adjacency_matrix_kernel(float* initial_rank, float* A, const int total_num, const int topk, const int nthreads, const int all_num) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < all_num; i += stride) {
        int ii = i / topk;
        A[ii * total_num + int(initial_rank[i])] = float(1.0);
    }
}

at::Tensor build_adjacency_matrix_forward(at::Tensor initial_rank) {
    const auto total_num = initial_rank.size(0);
    const auto topk = initial_rank.size(1);
    const auto all_num = total_num * topk;
    auto A = torch::zeros({total_num, total_num}, at::device(initial_rank.device()).dtype(at::ScalarType::Float));

    const int threads = 1024;
    const int blocks = (all_num + threads - 1) / threads;

    build_adjacency_matrix_kernel<<<blocks, threads>>>(initial_rank.data_ptr<float>(), A.data_ptr<float>(), total_num, topk, threads, all_num);
    return A;

}
